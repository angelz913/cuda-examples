
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int a, int b, int *sum) {
    *sum = a + b;
}

int main() {
    int a = 5;
    int b = 10;
    int sum;
    int *dev_sum;
    // Allocate memory on device.
    hipMalloc((void**)&dev_sum, sizeof(int));
    add<<<1,1>>>(5, 10, dev_sum);
    // Access memory on a device from host code.
    hipMemcpy(&sum, dev_sum, sizeof(int), hipMemcpyDeviceToHost);
    printf("%d + %d = %d\n", a, b, sum);
    // Free memory on device.
    hipFree(dev_sum);
    return 0;
}