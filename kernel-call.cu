
#include <hip/hip_runtime.h>
#include <iostream>

// The __global qualifier specifies a function to be compiled to run
// on a device.
__global__ void kernel(void) {}

int main() {
    // The angle brackets denote arguments to pass to the runtime system.
    kernel<<<1,1>>>();
    return 0;
}