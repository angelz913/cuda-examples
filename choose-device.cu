
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    // Get the ID of the current CUDA device.
    int dev;
    hipGetDevice(&dev);
    printf("%d\n", dev);

    // Get the properties.
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, dev);
    printf("%d.%d\n", prop.major, prop.minor);

    // Choose a CUDA device with certian properties.
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 7;
    prop.minor = 5;
    hipChooseDevice(&dev, &prop);
    printf("%d\n", dev);

    hipSetDevice(dev);
}